#include "hip/hip_runtime.h"
#include "../BHaH_defines.h"
/**
 * Given Cartesian point (x,y,z), this function unshifts the grid back to the origin to output the corresponding
 * (xx0,xx1,xx2) and the "closest" (i0,i1,i2) for the given grid
 */
__host__ __device__ void Cart_to_xx_and_nearest_i0i1i2__rfm__SinhSymTP(const params_struct *restrict params, const REAL xCart[3], REAL xx[3],
                                                                       int Cart_to_i0i1i2[3]) {

  // Set (Cartx, Carty, Cartz) relative to the global (as opposed to local) grid.
  //   This local grid may be offset from the origin by adjusting
  //   (Cart_originx, Cart_originy, Cart_originz) to nonzero values.
  REAL Cartx = xCart[0];
  REAL Carty = xCart[1];
  REAL Cartz = xCart[2];

  // Set the origin, (Cartx, Carty, Cartz) = (0, 0, 0), to the center of the local grid patch.
  Cartx -= params->Cart_originx;
  Carty -= params->Cart_originy;
  Cartz -= params->Cart_originz;
  {
    /*
     *  Original SymPy expressions:
     *  "[xx[0] = params->SQRT1_2*sqrt(Cartx**2 + Carty**2 + Cartz**2 - params->bScale**2 + sqrt(-4*Cartz**2*params->bScale**2 + params->bScale**4 +
     * 2*params->bScale**2*(Cartx**2 + Carty**2 + Cartz**2) + (Cartx**2 + Carty**2 + Cartz**2)**2))]"
     *  "[xx[1] = acos(params->SQRT1_2*sqrt(1 + (Cartx**2 + Carty**2 + Cartz**2)/params->bScale**2 - sqrt(-4*Cartz**2*params->bScale**2 +
     * params->bScale**4 + 2*params->bScale**2*(Cartx**2 + Carty**2 + Cartz**2) + (Cartx**2 + Carty**2 +
     * Cartz**2)**2)/params->bScale**2)*sign(Cartz))]"
     *  "[xx[2] = atan2(Carty, Cartx)]"
     */
    const REAL tmp1 = ((params->bScale) * (params->bScale));
    const REAL tmp2 = ((Cartx) * (Cartx)) + ((Carty) * (Carty)) + ((Cartz) * (Cartz));
    const REAL tmp4 = (1.0 / (tmp1));
    const REAL tmp3 = sqrt(-4 * ((Cartz) * (Cartz)) * tmp1 + ((params->bScale) * (params->bScale) * (params->bScale) * (params->bScale)) +
                           2 * tmp1 * tmp2 + ((tmp2) * (tmp2)));
    xx[0] = params->SQRT1_2 * sqrt(-tmp1 + tmp2 + tmp3);
    xx[1] = acos(params->SQRT1_2 * sqrt(tmp2 * tmp4 - tmp3 * tmp4 + 1) * (((Cartz) > 0) - ((Cartz) < 0)));
    xx[2] = atan2(Carty, Cartx);

    // Find the nearest grid indices (i0, i1, i2) for the given Cartesian coordinates (x, y, z).
    // Assuming a cell-centered grid, which follows the pattern:
    //   xx0[i0] = params->xxmin0 + ((REAL)(i0 - NGHOSTS) + 0.5) * params->dxx0
    // The index i0 can be derived as:
    //   i0 = (xx0[i0] - params->xxmin0) / params->dxx0 - 0.5 + NGHOSTS
    // Now, including typecasts:
    //   i0 = (int)((xx[0] - params->xxmin0) / params->dxx0 - 0.5 + (REAL)NGHOSTS)
    // The (int) typecast always rounds down, so we add 0.5 inside the outer parenthesis:
    //   i0 = (int)((xx[0] - params->xxmin0) / params->dxx0 - 0.5 + (REAL)NGHOSTS + 0.5)
    // The 0.5 values cancel out:
    //   i0 =           (int)( ( xx[0] - params->xxmin0 ) / params->dxx0 + (REAL)NGHOSTS )
    Cart_to_i0i1i2[0] = (int)((xx[0] - params->xxmin0) / params->dxx0 + (REAL)NGHOSTS);
    Cart_to_i0i1i2[1] = (int)((xx[1] - params->xxmin1) / params->dxx1 + (REAL)NGHOSTS);
    Cart_to_i0i1i2[2] = (int)((xx[2] - params->xxmin2) / params->dxx2 + (REAL)NGHOSTS);
  }
} // END FUNCTION Cart_to_xx_and_nearest_i0i1i2__rfm__SinhSymTP
