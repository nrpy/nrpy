#include "hip/hip_runtime.h"
#include "../BHaH_defines.h"
#include "../BHaH_function_prototypes.h"
/**
 * Kernel: compute_L2_norm_of_gridfunction_gpu.
 * Kernel to compute L2 quantities pointwise (not summed).
 */
__global__ static void compute_L2_norm_of_gridfunction_gpu(const size_t streamid, const REAL *restrict x0, const REAL *restrict x1,
                                                           const REAL *restrict x2, const REAL *restrict in_gfs, REAL *restrict aux_gfs,
                                                           const REAL integration_radius, const int gf_index) {
  MAYBE_UNUSED const int Nxx_plus_2NGHOSTS0 = d_params[streamid].Nxx_plus_2NGHOSTS0;
  MAYBE_UNUSED const int Nxx_plus_2NGHOSTS1 = d_params[streamid].Nxx_plus_2NGHOSTS1;
  MAYBE_UNUSED const int Nxx_plus_2NGHOSTS2 = d_params[streamid].Nxx_plus_2NGHOSTS2;

  MAYBE_UNUSED const REAL invdxx0 = d_params[streamid].invdxx0;
  MAYBE_UNUSED const REAL invdxx1 = d_params[streamid].invdxx1;
  MAYBE_UNUSED const REAL invdxx2 = d_params[streamid].invdxx2;

  MAYBE_UNUSED const int tid0 = blockIdx.x * blockDim.x + threadIdx.x;
  MAYBE_UNUSED const int tid1 = blockIdx.y * blockDim.y + threadIdx.y;
  MAYBE_UNUSED const int tid2 = blockIdx.z * blockDim.z + threadIdx.z;

  MAYBE_UNUSED const int stride0 = blockDim.x * gridDim.x;
  MAYBE_UNUSED const int stride1 = blockDim.y * gridDim.y;
  MAYBE_UNUSED const int stride2 = blockDim.z * gridDim.z;

  // Load necessary parameters from params_struct
  const REAL AMPLRHO = d_params[streamid].AMPLRHO;
  const REAL AMPLZ = d_params[streamid].AMPLZ;
  const REAL SINHWRHO = d_params[streamid].SINHWRHO;
  const REAL SINHWZ = d_params[streamid].SINHWZ;
  const REAL rho_slope = d_params[streamid].rho_slope;
  const REAL z_slope = d_params[streamid].z_slope;
  const REAL dxx0 = d_params[streamid].dxx0;
  const REAL dxx1 = d_params[streamid].dxx1;
  const REAL dxx2 = d_params[streamid].dxx2;

  for (int i2 = tid2 + NGHOSTS; i2 < Nxx_plus_2NGHOSTS2 - NGHOSTS; i2 += stride2) {
    MAYBE_UNUSED const REAL xx2 = x2[i2];
    for (int i1 = tid1 + NGHOSTS; i1 < Nxx_plus_2NGHOSTS1 - NGHOSTS; i1 += stride1) {
      MAYBE_UNUSED const REAL xx1 = x1[i1];
      for (int i0 = tid0 + NGHOSTS; i0 < Nxx_plus_2NGHOSTS0 - NGHOSTS; i0 += stride0) {
        MAYBE_UNUSED const REAL xx0 = x0[i0];

        /*
         *  Original SymPy expressions:
         *  "[const DOUBLE r = sqrt((rho_slope*xx0 + xx0**2*(AMPLRHO - rho_slope)*(exp(xx0/SINHWRHO) - exp(-xx0/SINHWRHO))/(exp(1/SINHWRHO) -
         * exp(-1/SINHWRHO)))**2 + (xx2**2*(AMPLZ - z_slope)*(exp(xx2/SINHWZ) - exp(-xx2/SINHWZ))/(exp(1/SINHWZ) - exp(-1/SINHWZ)) +
         * xx2*z_slope)**2)]"
         *  "[const DOUBLE sqrtdetgamma = (rho_slope*xx0 + xx0**2*(AMPLRHO - rho_slope)*(exp(xx0/SINHWRHO) - exp(-xx0/SINHWRHO))/(exp(1/SINHWRHO) -
         * exp(-1/SINHWRHO)))**2*(rho_slope + xx0**2*(AMPLRHO - rho_slope)*(exp(xx0/SINHWRHO)/SINHWRHO + exp(-xx0/SINHWRHO)/SINHWRHO)/(exp(1/SINHWRHO)
         * - exp(-1/SINHWRHO)) + 2*xx0*(AMPLRHO - rho_slope)*(exp(xx0/SINHWRHO) - exp(-xx0/SINHWRHO))/(exp(1/SINHWRHO) -
         * exp(-1/SINHWRHO)))**2*(xx2**2*(AMPLZ - z_slope)*(exp(xx2/SINHWZ)/SINHWZ + exp(-xx2/SINHWZ)/SINHWZ)/(exp(1/SINHWZ) - exp(-1/SINHWZ)) +
         * 2*xx2*(AMPLZ - z_slope)*(exp(xx2/SINHWZ) - exp(-xx2/SINHWZ))/(exp(1/SINHWZ) - exp(-1/SINHWZ)) + z_slope)**2]"
         */
        const REAL tmp0 = (1.0 / (SINHWRHO));
        const REAL tmp6 = AMPLRHO - rho_slope;
        const REAL tmp9 = (1.0 / (SINHWZ));
        const REAL tmp15 = AMPLZ - z_slope;
        const REAL tmp5 = (1.0 / (exp(tmp0) - exp(-tmp0)));
        const REAL tmp14 = (1.0 / (exp(tmp9) - exp(-tmp9)));
        const REAL tmp2 = exp(tmp0 * xx0);
        const REAL tmp3 = exp(-tmp0 * xx0);
        const REAL tmp7 = tmp5 * tmp6 * ((xx0) * (xx0));
        const REAL tmp11 = exp(tmp9 * xx2);
        const REAL tmp12 = exp(-tmp9 * xx2);
        const REAL tmp16 = tmp14 * tmp15 * ((xx2) * (xx2));
        const REAL tmp4 = tmp2 - tmp3;
        const REAL tmp13 = tmp11 - tmp12;
        const REAL tmp8 = ((rho_slope * xx0 + tmp4 * tmp7) * (rho_slope * xx0 + tmp4 * tmp7));
        const DOUBLE r = sqrt(tmp8 + ((tmp13 * tmp16 + xx2 * z_slope) * (tmp13 * tmp16 + xx2 * z_slope)));
        const DOUBLE sqrtdetgamma = tmp8 *
                                    ((rho_slope + 2 * tmp4 * tmp5 * tmp6 * xx0 + tmp7 * (tmp0 * tmp2 + tmp0 * tmp3)) *
                                     (rho_slope + 2 * tmp4 * tmp5 * tmp6 * xx0 + tmp7 * (tmp0 * tmp2 + tmp0 * tmp3))) *
                                    ((2 * tmp13 * tmp14 * tmp15 * xx2 + tmp16 * (tmp11 * tmp9 + tmp12 * tmp9) + z_slope) *
                                     (2 * tmp13 * tmp14 * tmp15 * xx2 + tmp16 * (tmp11 * tmp9 + tmp12 * tmp9) + z_slope));

        if (r < integration_radius) {
          const DOUBLE gf_of_x = in_gfs[IDX4(gf_index, i0, i1, i2)];
          const DOUBLE dV = sqrtdetgamma * dxx0 * dxx1 * dxx2;

          aux_gfs[IDX4(L2_SQUARED_DVGF, i0, i1, i2)] = gf_of_x * gf_of_x * dV;
          aux_gfs[IDX4(L2_DVGF, i0, i1, i2)] = dV;
        } // END if(r < integration_radius)

      } // END LOOP: for (int i0 = tid0+NGHOSTS; i0 < Nxx_plus_2NGHOSTS0 - NGHOSTS; i0 += stride0)
    } // END LOOP: for (int i1 = tid1+NGHOSTS; i1 < Nxx_plus_2NGHOSTS1 - NGHOSTS; i1 += stride1)
  } // END LOOP: for (int i2 = tid2+NGHOSTS; i2 < Nxx_plus_2NGHOSTS2 - NGHOSTS; i2 += stride2)
} // END FUNCTION compute_L2_norm_of_gridfunction_gpu

/**
 * Compute l2-norm of a gridfunction assuming a single grid.
 */
void compute_L2_norm_of_gridfunction__rfm__SinhCylindricalv2n2(commondata_struct *restrict commondata, params_struct *restrict params,
                                                               REAL *restrict xx[3], const REAL integration_radius, const int gf_index, REAL *l2norm,
                                                               const REAL *restrict in_gfs, REAL *restrict aux_gfs) {
#include "../set_CodeParameters.h"

  MAYBE_UNUSED const int Nxx_plus_2NGHOSTS_tot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2;
  REAL *restrict x0 = xx[0];
  REAL *restrict x1 = xx[1];
  REAL *restrict x2 = xx[2];

  // Since we're performing sums, make sure arrays are zero'd
  hipMemset(aux_gfs, 0, sizeof(REAL) * NUM_EVOL_GFS * Nxx_plus_2NGHOSTS_tot);

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_NELL_GRIDL2;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_NELL_GRIDL2;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_NELL_GRIDL2;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((params->Nxx_plus_2NGHOSTS0 + threads_in_x_dir - 1) / threads_in_x_dir,
                       (params->Nxx_plus_2NGHOSTS1 + threads_in_y_dir - 1) / threads_in_y_dir,
                       (params->Nxx_plus_2NGHOSTS2 + threads_in_z_dir - 1) / threads_in_z_dir);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  compute_L2_norm_of_gridfunction_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, x0, x1, x2, in_gfs, aux_gfs,
                                                                                                     integration_radius, gf_index);
  cudaCheckErrors(cudaKernel, "compute_L2_norm_of_gridfunction_gpu failure");

  // Set summation variables to compute l2-norm
  REAL squared_sum = find_global__sum(&aux_gfs[IDX4(L2_SQUARED_DVGF, 0, 0, 0)], Nxx_plus_2NGHOSTS_tot);
  REAL volume_sum = find_global__sum(&aux_gfs[IDX4(L2_DVGF, 0, 0, 0)], Nxx_plus_2NGHOSTS_tot);
  // Compute and output the log of the l2-norm.
  REAL local_norm = log10(1e-16 + sqrt(squared_sum / volume_sum)); // 1e-16 + ... avoids log10(0)

  // Compute and output the log of the l2-norm.
  *l2norm = log10(1e-16 + sqrt(squared_sum / volume_sum)); // 1e-16 + ... avoids log10(0)
} // END FUNCTION compute_L2_norm_of_gridfunction__rfm__SinhCylindricalv2n2
