#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_function_prototypes.h"
#include "intrinsics/cuda_intrinsics.h"

#define LOOP_ALL_GFS_GPS(ii)                                                                                                                         \
  const int tid0 = threadIdx.x + blockIdx.x * blockDim.x;                                                                                            \
  const int stride0 = blockDim.x * gridDim.x;                                                                                                        \
  for (int(ii) = (tid0);                                                                                                                             \
       (ii) < d_params[streamid].Nxx_plus_2NGHOSTS0 * d_params[streamid].Nxx_plus_2NGHOSTS1 * d_params[streamid].Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;  \
       (ii) += (stride0))
/**
 * Kernel: rk_substep_1_gpu.
 * Compute RK substep 1.
 */
__global__ static void rk_substep_1_gpu(const size_t streamid, REAL *restrict k_odd_gfs, REAL *restrict y_n_gfs,
                                        REAL *restrict y_nplus1_running_total_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k_odd_gfsL = k_odd_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static const double dblRK_Rational_1_2 = 1.0 / 2.0;
    const REAL_CUDA_ARRAY RK_Rational_1_2 = ConstCUDA(dblRK_Rational_1_2);

    static const double dblRK_Rational_1_6 = 1.0 / 6.0;
    const REAL_CUDA_ARRAY RK_Rational_1_6 = ConstCUDA(dblRK_Rational_1_6);

    const REAL_CUDA_ARRAY __rk_exp_0 = MulCUDA(RK_Rational_1_6, MulCUDA(dt, k_odd_gfsL));
    const REAL_CUDA_ARRAY __rk_exp_1 = FusedMulAddCUDA(RK_Rational_1_2, MulCUDA(dt, k_odd_gfsL), y_n_gfsL);
    WriteCUDA(&y_nplus1_running_total_gfs[i], __rk_exp_0);
    WriteCUDA(&k_odd_gfs[i], __rk_exp_1);
  }
} // END FUNCTION rk_substep_1_gpu

/**
 * Runge-Kutta function for substep 1.
 */
static void rk_substep_1__launcher(params_struct *restrict params, REAL *restrict k_odd_gfs, REAL *restrict y_n_gfs,
                                   REAL *restrict y_nplus1_running_total_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  {

    const size_t threads_in_x_dir = 32;
    const size_t threads_in_y_dir = 1;
    const size_t threads_in_z_dir = 1;
    dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
    dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
    size_t sm = 0;
    size_t streamid = params->grid_idx % NUM_STREAMS;
    rk_substep_1_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k_odd_gfs, y_n_gfs, y_nplus1_running_total_gfs, dt);
    cudaCheckErrors(cudaKernel, "rk_substep_1_gpu failure");
  }
} // END FUNCTION rk_substep_1__launcher

/**
 * Kernel: rk_substep_2_gpu.
 * Compute RK substep 2.
 */
__global__ static void rk_substep_2_gpu(const size_t streamid, REAL *restrict k_even_gfs, REAL *restrict y_nplus1_running_total_gfs,
                                        REAL *restrict y_n_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k_even_gfsL = k_even_gfs[i];
    const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static const double dblRK_Rational_1_2 = 1.0 / 2.0;
    const REAL_CUDA_ARRAY RK_Rational_1_2 = ConstCUDA(dblRK_Rational_1_2);

    static const double dblRK_Rational_1_3 = 1.0 / 3.0;
    const REAL_CUDA_ARRAY RK_Rational_1_3 = ConstCUDA(dblRK_Rational_1_3);

    const REAL_CUDA_ARRAY __rk_exp_0 = FusedMulAddCUDA(RK_Rational_1_3, MulCUDA(dt, k_even_gfsL), y_nplus1_running_total_gfsL);
    const REAL_CUDA_ARRAY __rk_exp_1 = FusedMulAddCUDA(RK_Rational_1_2, MulCUDA(dt, k_even_gfsL), y_n_gfsL);
    WriteCUDA(&y_nplus1_running_total_gfs[i], __rk_exp_0);
    WriteCUDA(&k_even_gfs[i], __rk_exp_1);
  }
} // END FUNCTION rk_substep_2_gpu

/**
 * Runge-Kutta function for substep 2.
 */
static void rk_substep_2__launcher(params_struct *restrict params, REAL *restrict k_even_gfs, REAL *restrict y_nplus1_running_total_gfs,
                                   REAL *restrict y_n_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  {

    const size_t threads_in_x_dir = 32;
    const size_t threads_in_y_dir = 1;
    const size_t threads_in_z_dir = 1;
    dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
    dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
    size_t sm = 0;
    size_t streamid = params->grid_idx % NUM_STREAMS;
    rk_substep_2_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k_even_gfs, y_nplus1_running_total_gfs, y_n_gfs, dt);
    cudaCheckErrors(cudaKernel, "rk_substep_2_gpu failure");
  }
} // END FUNCTION rk_substep_2__launcher

/**
 * Kernel: rk_substep_3_gpu.
 * Compute RK substep 3.
 */
__global__ static void rk_substep_3_gpu(const size_t streamid, REAL *restrict k_odd_gfs, REAL *restrict y_nplus1_running_total_gfs,
                                        REAL *restrict y_n_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k_odd_gfsL = k_odd_gfs[i];
    const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static const double dblRK_Rational_1_3 = 1.0 / 3.0;
    const REAL_CUDA_ARRAY RK_Rational_1_3 = ConstCUDA(dblRK_Rational_1_3);

    const REAL_CUDA_ARRAY __rk_exp_0 = FusedMulAddCUDA(RK_Rational_1_3, MulCUDA(dt, k_odd_gfsL), y_nplus1_running_total_gfsL);
    const REAL_CUDA_ARRAY __rk_exp_1 = FusedMulAddCUDA(dt, k_odd_gfsL, y_n_gfsL);
    WriteCUDA(&y_nplus1_running_total_gfs[i], __rk_exp_0);
    WriteCUDA(&k_odd_gfs[i], __rk_exp_1);
  }
} // END FUNCTION rk_substep_3_gpu

/**
 * Runge-Kutta function for substep 3.
 */
static void rk_substep_3__launcher(params_struct *restrict params, REAL *restrict k_odd_gfs, REAL *restrict y_nplus1_running_total_gfs,
                                   REAL *restrict y_n_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  {

    const size_t threads_in_x_dir = 32;
    const size_t threads_in_y_dir = 1;
    const size_t threads_in_z_dir = 1;
    dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
    dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
    size_t sm = 0;
    size_t streamid = params->grid_idx % NUM_STREAMS;
    rk_substep_3_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k_odd_gfs, y_nplus1_running_total_gfs, y_n_gfs, dt);
    cudaCheckErrors(cudaKernel, "rk_substep_3_gpu failure");
  }
} // END FUNCTION rk_substep_3__launcher

/**
 * Kernel: rk_substep_4_gpu.
 * Compute RK substep 4.
 */
__global__ static void rk_substep_4_gpu(const size_t streamid, REAL *restrict k_even_gfs, REAL *restrict y_n_gfs,
                                        REAL *restrict y_nplus1_running_total_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k_even_gfsL = k_even_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    const REAL y_nplus1_running_total_gfsL = y_nplus1_running_total_gfs[i];
    static const double dblRK_Rational_1_6 = 1.0 / 6.0;
    const REAL_CUDA_ARRAY RK_Rational_1_6 = ConstCUDA(dblRK_Rational_1_6);

    const REAL_CUDA_ARRAY __rk_exp_0 = AddCUDA(y_n_gfsL, FusedMulAddCUDA(RK_Rational_1_6, MulCUDA(dt, k_even_gfsL), y_nplus1_running_total_gfsL));
    WriteCUDA(&y_n_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_4_gpu

/**
 * Runge-Kutta function for substep 4.
 */
static void rk_substep_4__launcher(params_struct *restrict params, REAL *restrict k_even_gfs, REAL *restrict y_n_gfs,
                                   REAL *restrict y_nplus1_running_total_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  {

    const size_t threads_in_x_dir = 32;
    const size_t threads_in_y_dir = 1;
    const size_t threads_in_z_dir = 1;
    dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
    dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
    size_t sm = 0;
    size_t streamid = params->grid_idx % NUM_STREAMS;
    rk_substep_4_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k_even_gfs, y_n_gfs, y_nplus1_running_total_gfs, dt);
    cudaCheckErrors(cudaKernel, "rk_substep_4_gpu failure");
  }
} // END FUNCTION rk_substep_4__launcher

/**
 * Method of Lines (MoL) for "RK4" method: Step forward one full timestep.
 *
 */
void MoL_step_forward_in_time(commondata_struct *restrict commondata, griddata_struct *restrict griddata) {

  // C code implementation of -={ RK4 }=- Method of Lines timestepping.

  // First set the initial time:
  const REAL time_start = commondata->time;
  // -={ START k1 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 0.00000000000000000e+00 * commondata->dt;
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict y_nplus1_running_total_gfs = griddata[grid].gridfuncs.y_nplus1_running_total_gfs;
    MAYBE_UNUSED REAL *restrict k_odd_gfs = griddata[grid].gridfuncs.k_odd_gfs;
    MAYBE_UNUSED REAL *restrict k_even_gfs = griddata[grid].gridfuncs.k_even_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, y_n_gfs, k_odd_gfs);
    rk_substep_1__launcher(params, k_odd_gfs, y_n_gfs, y_nplus1_running_total_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, k_odd_gfs);
  }
  // -={ END k1 substep }=-

  // -={ START k2 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 5.00000000000000000e-01 * commondata->dt;
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict y_nplus1_running_total_gfs = griddata[grid].gridfuncs.y_nplus1_running_total_gfs;
    MAYBE_UNUSED REAL *restrict k_odd_gfs = griddata[grid].gridfuncs.k_odd_gfs;
    MAYBE_UNUSED REAL *restrict k_even_gfs = griddata[grid].gridfuncs.k_even_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, k_odd_gfs, k_even_gfs);
    rk_substep_2__launcher(params, k_even_gfs, y_nplus1_running_total_gfs, y_n_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, k_even_gfs);
  }
  // -={ END k2 substep }=-

  // -={ START k3 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 5.00000000000000000e-01 * commondata->dt;
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict y_nplus1_running_total_gfs = griddata[grid].gridfuncs.y_nplus1_running_total_gfs;
    MAYBE_UNUSED REAL *restrict k_odd_gfs = griddata[grid].gridfuncs.k_odd_gfs;
    MAYBE_UNUSED REAL *restrict k_even_gfs = griddata[grid].gridfuncs.k_even_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, k_even_gfs, k_odd_gfs);
    rk_substep_3__launcher(params, k_odd_gfs, y_nplus1_running_total_gfs, y_n_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, k_odd_gfs);
  }
  // -={ END k3 substep }=-

  // -={ START k4 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 1.00000000000000000e+00 * commondata->dt;
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict y_nplus1_running_total_gfs = griddata[grid].gridfuncs.y_nplus1_running_total_gfs;
    MAYBE_UNUSED REAL *restrict k_odd_gfs = griddata[grid].gridfuncs.k_odd_gfs;
    MAYBE_UNUSED REAL *restrict k_even_gfs = griddata[grid].gridfuncs.k_even_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, k_odd_gfs, k_even_gfs);
    rk_substep_4__launcher(params, k_even_gfs, y_n_gfs, y_nplus1_running_total_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, y_n_gfs);
  }
  // -={ END k4 substep }=-

  // Adding dt to commondata->time many times will induce roundoff error,
  // so here we set time based on the iteration number:
  commondata->time = (REAL)(commondata->nn + 1) * commondata->dt;

  // Increment the timestep n:
  commondata->nn++;
} // END FUNCTION MoL_step_forward_in_time
