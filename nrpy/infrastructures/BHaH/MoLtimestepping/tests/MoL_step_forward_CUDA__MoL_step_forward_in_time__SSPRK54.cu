#include "hip/hip_runtime.h"
#include "BHaH_defines.h"
#include "BHaH_function_prototypes.h"
#include "intrinsics/cuda_intrinsics.h"

#define LOOP_ALL_GFS_GPS(ii)                                                                                                                         \
  const int tid0 = threadIdx.x + blockIdx.x * blockDim.x;                                                                                            \
  const int stride0 = blockDim.x * gridDim.x;                                                                                                        \
  for (int(ii) = (tid0);                                                                                                                             \
       (ii) < d_params[streamid].Nxx_plus_2NGHOSTS0 * d_params[streamid].Nxx_plus_2NGHOSTS1 * d_params[streamid].Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;  \
       (ii) += (stride0))
/**
 * Kernel: rk_substep_1_gpu.
 * Compute RK substep 1.
 */
__global__ static void rk_substep_1_gpu(const size_t streamid, REAL *restrict k1_gfs, REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs,
                                        const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k1_gfsL = k1_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static constexpr double dblRK_Rational_27567112243069_70368744177664 = 27567112243069.0 / 70368744177664.0;
    const REAL_CUDA_ARRAY RK_Rational_27567112243069_70368744177664 = ConstCUDA(dblRK_Rational_27567112243069_70368744177664);

    const REAL_CUDA_ARRAY __rk_exp_0 = FusedMulAddCUDA(RK_Rational_27567112243069_70368744177664, MulCUDA(k1_gfsL, dt), y_n_gfsL);
    WriteCUDA(&next_y_input_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_1_gpu

/**
 * Runge-Kutta function for substep 1.
 */
static void rk_substep_1__launcher(params_struct *restrict params, REAL *restrict k1_gfs, REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs,
                                   const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_MOL_SUBSTEP;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_MOL_SUBSTEP;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_MOL_SUBSTEP;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  rk_substep_1_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k1_gfs, y_n_gfs, next_y_input_gfs, dt);
  cudaCheckErrors(cudaKernel, "rk_substep_1_gpu failure");
} // END FUNCTION rk_substep_1__launcher

/**
 * Kernel: rk_substep_2_gpu.
 * Compute RK substep 2.
 */
__global__ static void rk_substep_2_gpu(const size_t streamid, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict y_n_gfs,
                                        REAL *restrict next_y_input_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k1_gfsL = k1_gfs[i];
    const REAL k2_gfsL = k2_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static constexpr double dblRK_Rational_1659173807685965_4503599627370496 = 1659173807685965.0 / 4503599627370496.0;
    const REAL_CUDA_ARRAY RK_Rational_1659173807685965_4503599627370496 = ConstCUDA(dblRK_Rational_1659173807685965_4503599627370496);

    static constexpr double dblRK_Rational_7842355689270681_36028797018963968 = 7842355689270681.0 / 3.602879701896397e+16;
    const REAL_CUDA_ARRAY RK_Rational_7842355689270681_36028797018963968 = ConstCUDA(dblRK_Rational_7842355689270681_36028797018963968);

    const REAL_CUDA_ARRAY __rk_exp_0 =
        FusedMulAddCUDA(RK_Rational_1659173807685965_4503599627370496, MulCUDA(k2_gfsL, dt),
                        FusedMulAddCUDA(RK_Rational_7842355689270681_36028797018963968, MulCUDA(k1_gfsL, dt), y_n_gfsL));
    WriteCUDA(&next_y_input_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_2_gpu

/**
 * Runge-Kutta function for substep 2.
 */
static void rk_substep_2__launcher(params_struct *restrict params, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict y_n_gfs,
                                   REAL *restrict next_y_input_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_MOL_SUBSTEP;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_MOL_SUBSTEP;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_MOL_SUBSTEP;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  rk_substep_2_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k1_gfs, k2_gfs, y_n_gfs, next_y_input_gfs, dt);
  cudaCheckErrors(cudaKernel, "rk_substep_2_gpu failure");
} // END FUNCTION rk_substep_2__launcher

/**
 * Kernel: rk_substep_3_gpu.
 * Compute RK substep 3.
 */
__global__ static void rk_substep_3_gpu(const size_t streamid, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                        REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k1_gfsL = k1_gfs[i];
    const REAL k2_gfsL = k2_gfs[i];
    const REAL k3_gfsL = k3_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static constexpr double dblRK_Rational_2521268231078959_18014398509481984 = 2521268231078959.0 / 1.8014398509481984e+16;
    const REAL_CUDA_ARRAY RK_Rational_2521268231078959_18014398509481984 = ConstCUDA(dblRK_Rational_2521268231078959_18014398509481984);

    static constexpr double dblRK_Rational_4537678802552775_18014398509481984 = 4537678802552775.0 / 1.8014398509481984e+16;
    const REAL_CUDA_ARRAY RK_Rational_4537678802552775_18014398509481984 = ConstCUDA(dblRK_Rational_4537678802552775_18014398509481984);

    static constexpr double dblRK_Rational_5958592814262287_72057594037927936 = 5958592814262287.0 / 7.205759403792794e+16;
    const REAL_CUDA_ARRAY RK_Rational_5958592814262287_72057594037927936 = ConstCUDA(dblRK_Rational_5958592814262287_72057594037927936);

    const REAL_CUDA_ARRAY __rk_exp_0 =
        FusedMulAddCUDA(RK_Rational_4537678802552775_18014398509481984, MulCUDA(k3_gfsL, dt),
                        FusedMulAddCUDA(RK_Rational_5958592814262287_72057594037927936, MulCUDA(k1_gfsL, dt),
                                        FusedMulAddCUDA(RK_Rational_2521268231078959_18014398509481984, MulCUDA(k2_gfsL, dt), y_n_gfsL)));
    WriteCUDA(&next_y_input_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_3_gpu

/**
 * Runge-Kutta function for substep 3.
 */
static void rk_substep_3__launcher(params_struct *restrict params, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                   REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_MOL_SUBSTEP;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_MOL_SUBSTEP;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_MOL_SUBSTEP;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  rk_substep_3_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k1_gfs, k2_gfs, k3_gfs, y_n_gfs, next_y_input_gfs, dt);
  cudaCheckErrors(cudaKernel, "rk_substep_3_gpu failure");
} // END FUNCTION rk_substep_3__launcher

/**
 * Kernel: rk_substep_4_gpu.
 * Compute RK substep 4.
 */
__global__ static void rk_substep_4_gpu(const size_t streamid, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                        REAL *restrict k4_gfs, REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k1_gfsL = k1_gfs[i];
    const REAL k2_gfsL = k2_gfs[i];
    const REAL k3_gfsL = k3_gfs[i];
    const REAL k4_gfsL = k4_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static constexpr double dblRK_Rational_4144561800390297_36028797018963968 = 4144561800390297.0 / 3.602879701896397e+16;
    const REAL_CUDA_ARRAY RK_Rational_4144561800390297_36028797018963968 = ConstCUDA(dblRK_Rational_4144561800390297_36028797018963968);

    static constexpr double dblRK_Rational_4897486879351823_72057594037927936 = 4897486879351823.0 / 7.205759403792794e+16;
    const REAL_CUDA_ARRAY RK_Rational_4897486879351823_72057594037927936 = ConstCUDA(dblRK_Rational_4897486879351823_72057594037927936);

    static constexpr double dblRK_Rational_4908696163965517_9007199254740992 = 4908696163965517.0 / 9007199254740992.0;
    const REAL_CUDA_ARRAY RK_Rational_4908696163965517_9007199254740992 = ConstCUDA(dblRK_Rational_4908696163965517_9007199254740992);

    static constexpr double dblRK_Rational_7459218339277047_36028797018963968 = 7459218339277047.0 / 3.602879701896397e+16;
    const REAL_CUDA_ARRAY RK_Rational_7459218339277047_36028797018963968 = ConstCUDA(dblRK_Rational_7459218339277047_36028797018963968);

    const REAL_CUDA_ARRAY __rk_exp_0 = FusedMulAddCUDA(
        RK_Rational_4897486879351823_72057594037927936, MulCUDA(k1_gfsL, dt),
        FusedMulAddCUDA(RK_Rational_4908696163965517_9007199254740992, MulCUDA(k4_gfsL, dt),
                        FusedMulAddCUDA(RK_Rational_7459218339277047_36028797018963968, MulCUDA(k3_gfsL, dt),
                                        FusedMulAddCUDA(RK_Rational_4144561800390297_36028797018963968, MulCUDA(k2_gfsL, dt), y_n_gfsL))));
    WriteCUDA(&next_y_input_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_4_gpu

/**
 * Runge-Kutta function for substep 4.
 */
static void rk_substep_4__launcher(params_struct *restrict params, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                   REAL *restrict k4_gfs, REAL *restrict y_n_gfs, REAL *restrict next_y_input_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_MOL_SUBSTEP;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_MOL_SUBSTEP;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_MOL_SUBSTEP;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  rk_substep_4_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k1_gfs, k2_gfs, k3_gfs, k4_gfs, y_n_gfs, next_y_input_gfs,
                                                                                  dt);
  cudaCheckErrors(cudaKernel, "rk_substep_4_gpu failure");
} // END FUNCTION rk_substep_4__launcher

/**
 * Kernel: rk_substep_5_gpu.
 * Compute RK substep 5.
 */
__global__ static void rk_substep_5_gpu(const size_t streamid, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                        REAL *restrict k4_gfs, REAL *restrict k5_gfs, REAL *restrict y_n_gfs, const REAL dt) {
  LOOP_ALL_GFS_GPS(i) {
    const REAL k1_gfsL = k1_gfs[i];
    const REAL k2_gfsL = k2_gfs[i];
    const REAL k3_gfsL = k3_gfs[i];
    const REAL k4_gfsL = k4_gfs[i];
    const REAL k5_gfsL = k5_gfs[i];
    const REAL y_n_gfsL = y_n_gfs[i];
    static constexpr double dblRK_Rational_1235962931917479_4503599627370496 = 1235962931917479.0 / 4503599627370496.0;
    const REAL_CUDA_ARRAY RK_Rational_1235962931917479_4503599627370496 = ConstCUDA(dblRK_Rational_1235962931917479_4503599627370496);

    static constexpr double dblRK_Rational_2644727643550321_18014398509481984 = 2644727643550321.0 / 1.8014398509481984e+16;
    const REAL_CUDA_ARRAY RK_Rational_2644727643550321_18014398509481984 = ConstCUDA(dblRK_Rational_2644727643550321_18014398509481984);

    static constexpr double dblRK_Rational_3756320236709225_36028797018963968 = 3756320236709225.0 / 3.602879701896397e+16;
    const REAL_CUDA_ARRAY RK_Rational_3756320236709225_36028797018963968 = ConstCUDA(dblRK_Rational_3756320236709225_36028797018963968);

    static constexpr double dblRK_Rational_4476270149944963_18014398509481984 = 4476270149944963.0 / 1.8014398509481984e+16;
    const REAL_CUDA_ARRAY RK_Rational_4476270149944963_18014398509481984 = ConstCUDA(dblRK_Rational_4476270149944963_18014398509481984);

    static constexpr double dblRK_Rational_8142777736761735_36028797018963968 = 8142777736761735.0 / 3.602879701896397e+16;
    const REAL_CUDA_ARRAY RK_Rational_8142777736761735_36028797018963968 = ConstCUDA(dblRK_Rational_8142777736761735_36028797018963968);

    const REAL_CUDA_ARRAY __rk_exp_0 = FusedMulAddCUDA(
        RK_Rational_2644727643550321_18014398509481984, MulCUDA(k1_gfsL, dt),
        FusedMulAddCUDA(
            RK_Rational_3756320236709225_36028797018963968, MulCUDA(k3_gfsL, dt),
            FusedMulAddCUDA(RK_Rational_4476270149944963_18014398509481984, MulCUDA(k2_gfsL, dt),
                            FusedMulAddCUDA(RK_Rational_8142777736761735_36028797018963968, MulCUDA(k5_gfsL, dt),
                                            FusedMulAddCUDA(RK_Rational_1235962931917479_4503599627370496, MulCUDA(k4_gfsL, dt), y_n_gfsL)))));
    WriteCUDA(&y_n_gfs[i], __rk_exp_0);
  }
} // END FUNCTION rk_substep_5_gpu

/**
 * Runge-Kutta function for substep 5.
 */
static void rk_substep_5__launcher(params_struct *restrict params, REAL *restrict k1_gfs, REAL *restrict k2_gfs, REAL *restrict k3_gfs,
                                   REAL *restrict k4_gfs, REAL *restrict k5_gfs, REAL *restrict y_n_gfs, const REAL dt) {
  const int Nxx_plus_2NGHOSTS0 = params->Nxx_plus_2NGHOSTS0;
  const int Nxx_plus_2NGHOSTS1 = params->Nxx_plus_2NGHOSTS1;
  const int Nxx_plus_2NGHOSTS2 = params->Nxx_plus_2NGHOSTS2;
  MAYBE_UNUSED const int Ntot = Nxx_plus_2NGHOSTS0 * Nxx_plus_2NGHOSTS1 * Nxx_plus_2NGHOSTS2 * NUM_EVOL_GFS;

  const size_t threads_in_x_dir = BHAH_THREADS_IN_X_DIR_MOL_SUBSTEP;
  const size_t threads_in_y_dir = BHAH_THREADS_IN_Y_DIR_MOL_SUBSTEP;
  const size_t threads_in_z_dir = BHAH_THREADS_IN_Z_DIR_MOL_SUBSTEP;
  dim3 threads_per_block(threads_in_x_dir, threads_in_y_dir, threads_in_z_dir);
  dim3 blocks_per_grid((Ntot + threads_in_x_dir - 1) / threads_in_x_dir, 1, 1);
  size_t sm = 0;
  size_t streamid = params->grid_idx % NUM_STREAMS;
  rk_substep_5_gpu<<<blocks_per_grid, threads_per_block, sm, streams[streamid]>>>(streamid, k1_gfs, k2_gfs, k3_gfs, k4_gfs, k5_gfs, y_n_gfs, dt);
  cudaCheckErrors(cudaKernel, "rk_substep_5_gpu failure");
} // END FUNCTION rk_substep_5__launcher

/**
 * Method of Lines (MoL) for "SSPRK54" method: Step forward one full timestep.
 *
 */
void MoL_step_forward_in_time(commondata_struct *restrict commondata, griddata_struct *restrict griddata) {

  // C code implementation of -={ SSPRK54 }=- Method of Lines timestepping.

  // First set the initial time:
  const REAL time_start = commondata->time;
  // -={ START k1 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 0.00000000000000000e+00 * commondata->dt;
    cpyHosttoDevice_params__constant(&griddata[grid].params, griddata[grid].params.grid_idx % NUM_STREAMS);
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict next_y_input_gfs = griddata[grid].gridfuncs.next_y_input_gfs;
    MAYBE_UNUSED REAL *restrict k1_gfs = griddata[grid].gridfuncs.k1_gfs;
    MAYBE_UNUSED REAL *restrict k2_gfs = griddata[grid].gridfuncs.k2_gfs;
    MAYBE_UNUSED REAL *restrict k3_gfs = griddata[grid].gridfuncs.k3_gfs;
    MAYBE_UNUSED REAL *restrict k4_gfs = griddata[grid].gridfuncs.k4_gfs;
    MAYBE_UNUSED REAL *restrict k5_gfs = griddata[grid].gridfuncs.k5_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, y_n_gfs, k1_gfs);
    rk_substep_1__launcher(params, k1_gfs, y_n_gfs, next_y_input_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, next_y_input_gfs);
  }
  // -={ END k1 substep }=-

  // -={ START k2 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 3.91752227003919984e-01 * commondata->dt;
    cpyHosttoDevice_params__constant(&griddata[grid].params, griddata[grid].params.grid_idx % NUM_STREAMS);
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict next_y_input_gfs = griddata[grid].gridfuncs.next_y_input_gfs;
    MAYBE_UNUSED REAL *restrict k1_gfs = griddata[grid].gridfuncs.k1_gfs;
    MAYBE_UNUSED REAL *restrict k2_gfs = griddata[grid].gridfuncs.k2_gfs;
    MAYBE_UNUSED REAL *restrict k3_gfs = griddata[grid].gridfuncs.k3_gfs;
    MAYBE_UNUSED REAL *restrict k4_gfs = griddata[grid].gridfuncs.k4_gfs;
    MAYBE_UNUSED REAL *restrict k5_gfs = griddata[grid].gridfuncs.k5_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, next_y_input_gfs, k2_gfs);
    rk_substep_2__launcher(params, k1_gfs, k2_gfs, y_n_gfs, next_y_input_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, next_y_input_gfs);
  }
  // -={ END k2 substep }=-

  // -={ START k3 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 5.86079688967789947e-01 * commondata->dt;
    cpyHosttoDevice_params__constant(&griddata[grid].params, griddata[grid].params.grid_idx % NUM_STREAMS);
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict next_y_input_gfs = griddata[grid].gridfuncs.next_y_input_gfs;
    MAYBE_UNUSED REAL *restrict k1_gfs = griddata[grid].gridfuncs.k1_gfs;
    MAYBE_UNUSED REAL *restrict k2_gfs = griddata[grid].gridfuncs.k2_gfs;
    MAYBE_UNUSED REAL *restrict k3_gfs = griddata[grid].gridfuncs.k3_gfs;
    MAYBE_UNUSED REAL *restrict k4_gfs = griddata[grid].gridfuncs.k4_gfs;
    MAYBE_UNUSED REAL *restrict k5_gfs = griddata[grid].gridfuncs.k5_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, next_y_input_gfs, k3_gfs);
    rk_substep_3__launcher(params, k1_gfs, k2_gfs, k3_gfs, y_n_gfs, next_y_input_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, next_y_input_gfs);
  }
  // -={ END k3 substep }=-

  // -={ START k4 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 4.74542363026869973e-01 * commondata->dt;
    cpyHosttoDevice_params__constant(&griddata[grid].params, griddata[grid].params.grid_idx % NUM_STREAMS);
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict next_y_input_gfs = griddata[grid].gridfuncs.next_y_input_gfs;
    MAYBE_UNUSED REAL *restrict k1_gfs = griddata[grid].gridfuncs.k1_gfs;
    MAYBE_UNUSED REAL *restrict k2_gfs = griddata[grid].gridfuncs.k2_gfs;
    MAYBE_UNUSED REAL *restrict k3_gfs = griddata[grid].gridfuncs.k3_gfs;
    MAYBE_UNUSED REAL *restrict k4_gfs = griddata[grid].gridfuncs.k4_gfs;
    MAYBE_UNUSED REAL *restrict k5_gfs = griddata[grid].gridfuncs.k5_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, next_y_input_gfs, k4_gfs);
    rk_substep_4__launcher(params, k1_gfs, k2_gfs, k3_gfs, k4_gfs, y_n_gfs, next_y_input_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, next_y_input_gfs);
  }
  // -={ END k4 substep }=-

  // -={ START k5 substep }=-
  for (int grid = 0; grid < commondata->NUMGRIDS; grid++) {
    commondata->time = time_start + 9.35010631009239979e-01 * commondata->dt;
    cpyHosttoDevice_params__constant(&griddata[grid].params, griddata[grid].params.grid_idx % NUM_STREAMS);
    // Set gridfunction aliases, from griddata[].gridfuncs.
    MAYBE_UNUSED REAL *restrict y_n_gfs = griddata[grid].gridfuncs.y_n_gfs;
    MAYBE_UNUSED REAL *restrict next_y_input_gfs = griddata[grid].gridfuncs.next_y_input_gfs;
    MAYBE_UNUSED REAL *restrict k1_gfs = griddata[grid].gridfuncs.k1_gfs;
    MAYBE_UNUSED REAL *restrict k2_gfs = griddata[grid].gridfuncs.k2_gfs;
    MAYBE_UNUSED REAL *restrict k3_gfs = griddata[grid].gridfuncs.k3_gfs;
    MAYBE_UNUSED REAL *restrict k4_gfs = griddata[grid].gridfuncs.k4_gfs;
    MAYBE_UNUSED REAL *restrict k5_gfs = griddata[grid].gridfuncs.k5_gfs;
    MAYBE_UNUSED REAL *restrict auxevol_gfs = griddata[grid].gridfuncs.auxevol_gfs;
    // Set pointers to this grid's params, rfm_struct/xx, bc_struct, etc.
    MAYBE_UNUSED params_struct *restrict params = &griddata[grid].params;
    MAYBE_UNUSED REAL *restrict xx[3];
    for (int ww = 0; ww < 3; ww++)
      xx[ww] = griddata[grid].xx[ww];
    rhs_eval(commondata, params, rfmstruct, auxevol_gfs, next_y_input_gfs, k5_gfs);
    rk_substep_5__launcher(params, k1_gfs, k2_gfs, k3_gfs, k4_gfs, k5_gfs, y_n_gfs, commondata->dt);
    if (strncmp(commondata->outer_bc_type, "extrapolation", 50) == 0)
      apply_bcs_outerextrap_and_inner(commondata, params, bcstruct, y_n_gfs);
  }
  // -={ END k5 substep }=-

  // Adding dt to commondata->time many times will induce roundoff error,
  // so here we set time based on the iteration number:
  commondata->time = (REAL)(commondata->nn + 1) * commondata->dt;

  // Increment the timestep n:
  commondata->nn++;
} // END FUNCTION MoL_step_forward_in_time
